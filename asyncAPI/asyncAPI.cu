#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

#define N (16 * 1024 * 1024)
#define VAL 26

__global__ void increment_kernel(int *g_data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + VAL;
}


bool correct_output(int *data) {
  for (int i = 0; i < N; i++)
    if (data[i] != VAL) {
      printf("Error! data[%d] = %d, ref = %d\n", i, data[i], VAL);
      return false;
    }

  return true;
}

int cpucheck(){
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    int *a = 0;
    int nbytes = N * sizeof(int);

    checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
    memset(a, 0, nbytes);

    for (auto i=0;i < N;i++){
            a[i] = VAL;
    }
    sdkStopTimer(&timer);
    printf("time spent by CPU in incrementing operations: %.2f ms\n", sdkGetTimerValue(&timer));
    
    bool bFinalResults = correct_output(a);
    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);  

 }

int main(int argc, char *argv[]) {

  int devID;
  hipDeviceProp_t deviceProps;

  printf("[%s] - Starting...\n", argv[0]);

  // This will pick the best possible CUDA capable device
  devID = findCudaDevice(argc, (const char **)argv);

    // get device name
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s]\n", deviceProps.name);

  int nbytes = N * sizeof(int);
 
  // allocate host memory  
  int *a = 0;
  checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
  memset(a, 0, nbytes);

  // allocate device memory
  int *d_a = 0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

    // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks = dim3(N / threads.x, 1);
  printf("Processing %d integers with below threads and blocks!!\n",N);
  printf("Threads Dimension (X,Y,Z) : %d,%d,%d\n",threads.x,threads.y,threads.z);
  printf("Blocks Dimension (X,Y,Z) : %d,%d,%d\n",blocks.x,blocks.y,blocks.z);

   hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);

  checkCudaErrors(hipDeviceSynchronize());
  float gpu_time = 0.0f;

  // asynchronously issue work to the GPU (all to stream 0)
  checkCudaErrors(hipProfilerStart());
  sdkStartTimer(&timer);
  hipEventRecord(start, 0);
  hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
  increment_kernel<<<blocks, threads, 0, 0>>>(d_a);
  hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
  hipEventRecord(stop, 0);
  sdkStopTimer(&timer);
  checkCudaErrors(hipProfilerStop());

    // have CPU do some work while waiting for stage 1 to finish
  unsigned long int counter = 0;

  while (hipEventQuery(stop) == hipErrorNotReady) {
    counter++;
  }

  checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
  printf("time spent executing by the GPU: %.2f ms\n", gpu_time);
  printf("time spent by CPU in CUDA calls: %.2f ms\n", sdkGetTimerValue(&timer));
  printf("CPU executed %lu iterations while waiting for GPU to finish\n",
         counter);

  // check the output for correctness
  bool bFinalResults = correct_output(a);

  // release resources
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipHostFree(a));
  checkCudaErrors(hipFree(d_a));
 
  cpucheck();
  exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);  
  //return 0;

}